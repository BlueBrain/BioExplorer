#include "hip/hip_runtime.h"
/* Copyright (c) 2019, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 *
 * This file is part of Brayns <https://github.com/BlueBrain/Brayns>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include "TransferFunction.cuh"

#include <optix_world.h>

struct PerRayData_radiance
{
    float3 result;
    float importance;
    int depth;
};

// Scene
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );

// Material attributes
rtDeclareVariable(float3, Kd, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

// Textures
rtDeclareVariable(int, albedoMetallic_map, , );
rtDeclareVariable(float2, texcoord, attribute texcoord, );

// Simulation data
rtBuffer<float> simulation_data;
rtDeclareVariable(unsigned long, simulation_idx, attribute simulation_idx, );

// Transfer function
rtBuffer<float3> tfColors;
rtBuffer<float> tfOpacities;
rtDeclareVariable(float, tfMinValue, , );
rtDeclareVariable(float, tfRange, , );
rtDeclareVariable(uint, tfSize, , );

// Rendering
rtDeclareVariable(float, mainExposure, , );

static __device__ inline void shade(bool textured)
{
    float3 world_shading_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 world_geometric_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));

    float3 p_normal = optix::faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

    float3 p_Kd;
    if (textured)
        p_Kd = make_float3(optix::rtTex2D<float4>(albedoMetallic_map, texcoord.x, texcoord.y));
    else
        p_Kd = Kd;

    if (simulation_data.size() > 0)
    {
        const float4 userDataColor = calcTransferFunctionColor(tfMinValue, tfMinValue + tfRange,
                                                               simulation_data[simulation_idx], tfColors, tfOpacities);
        p_Kd = p_Kd * (1.f - userDataColor.w) + make_float3(userDataColor) * userDataColor.w;
    }

    prd.result = mainExposure * p_Kd * max(0.f, optix::dot(-ray.direction, p_normal));
}

RT_PROGRAM void any_hit_shadow()
{
    rtTerminateRay();
}

RT_PROGRAM void closest_hit_radiance()
{
    shade(false);
}

RT_PROGRAM void closest_hit_radiance_textured()
{
    shade(true);
}
