#include "hip/hip_runtime.h"
/* Copyright (c) 2020, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 * Responsible Author: Cyrille Favreau <cyrille.favreau@epfl.ch>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

// #include <optix.h>
// #include <optix_world.h>
// #include <optixu/optixu_math_namespace.h>

#include "../../CommonStructs.h"
#include "../Environment.h"
#include "../Helpers.h"
#include "../Random.h"
#include "TransferFunction.h"

#include <brayns/common/CommonTypes.h>

using namespace optix;

const float DEFAULT_VOLUME_SHADOW_THRESHOLD = 0.1f;

struct PerRayData_shadow
{
    float3 attenuation;
};

// System
rtDeclareVariable(float3, bad_color, , );

// Material attributes
rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float3, Kr, , );
rtDeclareVariable(float3, Ko, , );
rtDeclareVariable(float, glossiness, , );
rtDeclareVariable(float, refraction_index, , );
rtDeclareVariable(float, phong_exp, , );
rtDeclareVariable(uint, shading_mode, , );
rtDeclareVariable(float, user_parameter, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

// Textures
rtDeclareVariable(int, albedoMetallic_map, , );
rtDeclareVariable(float2, texcoord, attribute texcoord, );

// Scene
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );
rtDeclareVariable(unsigned int, frame, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(unsigned int, radianceRayType, , );
rtDeclareVariable(unsigned int, shadowRayType, , );

rtDeclareVariable(float, sceneEpsilon, , );
rtDeclareVariable(float, epsilonFactor, , );

rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float4, jitter4, , );

// Lights
rtBuffer<BasicLight> lights;
rtDeclareVariable(float3, ambientLightColor, , );

// Volume
rtBuffer<unsigned char> volumeData;
rtDeclareVariable(uint3, volumeDimensions, , );
rtDeclareVariable(float3, volumeOffset, , );
rtDeclareVariable(float3, volumeElementSpacing, , );
rtDeclareVariable(uint, volumeSamplesPerRay, , );
rtDeclareVariable(uint, volumeDataTypeSize, , );
// Volume shading
rtDeclareVariable(uint, volumeGradientShadingEnabled, , );
rtDeclareVariable(float, volumeAdaptiveMaxSamplingRate, , );
rtDeclareVariable(uint, volumeAdaptiveSampling, , );
rtDeclareVariable(uint, volumeSingleShade, , );
rtDeclareVariable(uint, volumePreIntegration, , );
rtDeclareVariable(float, volumeSamplingRate, , );
rtDeclareVariable(float3, volumeSpecular, , );

// Simulation data
rtBuffer<float> simulation_data;
rtDeclareVariable(unsigned long, simulation_idx, attribute simulation_idx, );

// Transfer function
rtBuffer<float3> tfColors;
rtBuffer<float> tfOpacities;
rtDeclareVariable(float, tfMinValue, , );
rtDeclareVariable(float, tfRange, , );
rtDeclareVariable(uint, tfSize, , );

// Rendering
rtDeclareVariable(int, maxBounces, , );
rtDeclareVariable(float, shadows, , );
rtDeclareVariable(float, softShadows, , );
rtDeclareVariable(int, softShadowsSamples, , );
rtDeclareVariable(float, mainExposure, , );
rtDeclareVariable(float, giDistance, , );
rtDeclareVariable(float, giWeight, , );
rtDeclareVariable(int, giSamples, , );
rtDeclareVariable(unsigned int, matrixFilter, , );
rtDeclareVariable(float, fogStart, , );
rtDeclareVariable(float, fogThickness, , );

rtBuffer<uchar4, 2> output_buffer;

static __device__ inline bool volumeIntersection(const optix::Ray& ray, float& t0, float& t1)
{
    float3 boxmin = volumeOffset + make_float3(0.f);
    float3 boxmax = volumeOffset + make_float3(volumeDimensions) / volumeElementSpacing;

    float3 a = (boxmin - ray.origin) / ray.direction;
    float3 b = (boxmax - ray.origin) / ray.direction;
    float3 near = fminf(a, b);
    float3 far = fmaxf(a, b);
    t0 = fmaxf(near);
    t1 = fminf(far);

    return (t0 <= t1);
}

static __device__ inline float3 frac(const float3 x)
{
    return x - optix::floor(x);
}

static __device__ inline float mix(const float x, const float y, const float a)
{
    return x * (1.f - a) + y * a;
}

static __device__ inline float hash(float n)
{
    return frac(make_float3(sin(n + 1.951f) * 43758.5453f)).x;
}

static __device__ float noise(const float3& x)
{
    // hash based 3d value noise
    float3 p = optix::floor(x);
    float3 f = frac(x);

    f = f * f * (make_float3(3.0f) - make_float3(2.0f) * f);
    float n = p.x + p.y * 57.0f + 113.0f * p.z;
    return mix(mix(mix(hash(n + 0.0f), hash(n + 1.0f), f.x), mix(hash(n + 57.0f), hash(n + 58.0f), f.x), f.y),
               mix(mix(hash(n + 113.0f), hash(n + 114.0f), f.x), mix(hash(n + 170.0f), hash(n + 171.0f), f.x), f.y),
               f.z);
}

static __device__ inline float3 mod(const float3& v, const int m)
{
    return make_float3(v.x - m * floor(v.x / m), v.y - m * floor(v.y / m), v.z - m * floor(v.z / m));
}

static __device__ float cells(const float3& p, float cellCount)
{
    const float3 pCell = p * cellCount;
    float d = 1.0e10;
    for (int xo = -1; xo <= 1; xo++)
    {
        for (int yo = -1; yo <= 1; yo++)
        {
            for (int zo = -1; zo <= 1; zo++)
            {
                float3 tp = floor(pCell) + make_float3(xo, yo, zo);

                tp = pCell - tp - noise(mod(tp, cellCount / 1));

                d = min(d, optix::dot(tp, tp));
            }
        }
    }
    d = min(d, 1.0f);
    d = max(d, 0.0f);
    return d;
}

static __device__ float worleyNoise(const float3& p, float cellCount)
{
    return cells(p, cellCount);
}

static __device__ float3 refractedVector(const float3 direction, const float3 normal, const float n1, const float n2)
{
    if (n2 == 0.f)
        return direction;
    const float eta = n1 / n2;
    const float cos1 = -optix::dot(direction, normal);
    const float cos2 = 1.f - eta * eta * (1.f - cos1 * cos1);
    if (cos2 > 0.f)
        return ::optix::normalize(eta * direction + (eta * cos1 - sqrt(cos2)) * normal);
    return direction;
}

static __device__ void compose(const float4& src, float4& dst, const float alphaRatio = 1.0)
{
    const float a = alphaRatio * src.w;
    dst = make_float4((1.f - dst.w) * a * make_float3(src) + dst.w * make_float3(dst), dst.w + a);
}

static __device__ float getVoxelValue(const ulong index)
{
    float voxelValue;
    switch (volumeDataTypeSize)
    {
    case 2:
    {
        unsigned char a = volumeData[index * volumeDataTypeSize + 1];
        unsigned char b = volumeData[index * volumeDataTypeSize];
        voxelValue = a * 256 + b;
        break;
    }
    default:
    {
        voxelValue = volumeData[index];
        break;
    }
    }
    return voxelValue;
}

static __device__ float getVolumeShadowContribution(const optix::Ray& volumeRay)
{
    float shadowIntensity = 0.f;
    float t0, t1;
    if (!volumeIntersection(volumeRay, t0, t1))
        return shadowIntensity;

    t0 = max(0.f, t0);
    float tstep = volumeSamplingRate * 4.f;
    float t = t0 + tstep;

    while (t < t1 && shadowIntensity < 1.f)
    {
        const float3 point = volumeRay.origin + volumeRay.direction * t;
        if (point.x > 0.f && point.x < volumeDimensions.x && point.y > 0.f && point.y < volumeDimensions.y &&
            point.z > 0.f && point.z < volumeDimensions.z)
        {
            const ulong index = (long)((ulong)floor(point.x) + (ulong)floor(point.y) * volumeDimensions.x +
                                       (ulong)floor(point.z) * volumeDimensions.x * volumeDimensions.y);
            const float voxelValue = getVoxelValue(index);
            const float4 voxelColor =
                calcTransferFunctionColor(tfMinValue, tfMinValue + tfRange, voxelValue, tfColors, tfOpacities);
            shadowIntensity += voxelColor.w;
        }
        t += tstep;
    }
    return shadowIntensity;
}

static __device__ float4 getVolumeContribution(const optix::Ray& volumeRay)
{
    if (tfColors.size() == 0)
        return make_float4(0.f, 1.f, 0.f, 0.f);

    const uint nbSamples = 7;
    const float3 samples[nbSamples] = {{0, 0, 0}, {0, -1, 0}, {0, 1, 0}, {-1, 0, 0}, {1, 0, 0}, {0, 0, 1}, {0, 0, -1}};

    float4 pathColor = make_float4(0.f);

    float t0, t1;
    if (!volumeIntersection(volumeRay, t0, t1))
        return pathColor;

    optix::size_t2 screen = output_buffer.size();
    unsigned int seed = tea<16>(screen.x * launch_index.y + launch_index.x, frame);

    float t = max(0.f, t0);
    uint iteration = 0;
    while (t < (t1 - volumeSamplingRate) && pathColor.w < 1.f)
    {
        const float random = rnd(seed) * volumeSamplingRate;
        float4 voxelColor = make_float4(0.f);

        const uint nbSamplesToCompute =
            volumeGradientShadingEnabled ? (volumeSingleShade ? (iteration == 0 ? nbSamples : 1) : nbSamples) : 1;
        uint computedSamples = 0;
        uint computedShadowSamples = 0;
        float shadowIntensity = 0.f;
        for (int i = 0; i < nbSamplesToCompute; ++i)
        {
            const float3 point =
                ((volumeRay.origin + samples[i] * volumeSamplingRate + volumeRay.direction * (t + random)) -
                 volumeOffset) /
                volumeElementSpacing;

            if (point.x > 0.f && point.x < volumeDimensions.x && point.y > 0.f && point.y < volumeDimensions.y &&
                point.z > 0.f && point.z < volumeDimensions.z)
            {
                ++computedSamples;
                const ulong index = (long)((ulong)floor(point.x) + (ulong)floor(point.y) * volumeDimensions.x +
                                           (ulong)floor(point.z) * volumeDimensions.x * volumeDimensions.y);
                const float voxelValue = getVoxelValue(index);
                voxelColor +=
                    calcTransferFunctionColor(tfMinValue, tfMinValue + tfRange, voxelValue, tfColors, tfOpacities);

                // Determine light contribution
                if (computedShadowSamples == 0 && shadows > 0.f && voxelColor.w > DEFAULT_VOLUME_SHADOW_THRESHOLD)
                {
                    ++computedShadowSamples;
                    for (int i = 0; i < lights.size(); ++i)
                    {
                        BasicLight light = lights[i];
                        optix::Ray shadowRay = volumeRay;
                        switch (light.type)
                        {
                        case BASIC_LIGHT_TYPE_POINT:
                        {
                            // Point light
                            float3 lightPosition = light.pos;
                            if (softShadows > 0.f)
                                // Soft shadows
                                lightPosition +=
                                    softShadows * make_float3(rnd(seed) - 0.5f, rnd(seed) - 0.5f, rnd(seed) - 0.5f);
                            shadowRay.origin = lightPosition;
                            shadowRay.direction = optix::normalize(lightPosition - point);
                            break;
                        }
                        case BASIC_LIGHT_TYPE_DIRECTIONAL:
                        {
                            // Directional light
                            float3 lightDirection = light.dir;
                            if (softShadows > 0.f)
                                // Soft shadows
                                lightDirection +=
                                    softShadows * make_float3(rnd(seed) - 0.5f, rnd(seed) - 0.5f, rnd(seed) - 0.5f);
                            shadowRay.origin = point;
                            shadowRay.direction = optix::normalize(-1.f * lightDirection);
                            break;
                        }
                        }

                        shadowIntensity += getVolumeShadowContribution(shadowRay);
                    }
                }
            }
        }

        if (computedSamples > 0)
        {
            const float lightAttenuation = 1.f - shadows * shadowIntensity;
            voxelColor.x *= lightAttenuation;
            voxelColor.y *= lightAttenuation;
            voxelColor.z *= lightAttenuation;
            compose(voxelColor / float(computedSamples), pathColor);
        }
        t += volumeSamplingRate;
        ++iteration;
    }

    compose(make_float4(getEnvironmentColor(), 1.f - pathColor.w), pathColor);

    return ::optix::clamp(pathColor, 0.f, 1.f);
}

static __device__ void phongShadowed(float3 p_Ko)
{
    // this material is opaque, so it fully attenuates all shadow rays
    prd_shadow.attenuation = 1.f - p_Ko;
    rtTerminateRay();
}

static __device__ void phongShade(float3 p_Kd, float3 p_Ka, float3 p_Ks, float3 p_Kr, float3 p_Ko,
                                  float p_refractionIndex, float p_phong_exp, float p_glossiness,
                                  unsigned int p_shadingMode, float p_user_parameter, float3 p_normal)
{
    const float3 hit_point = ray.origin + t_hit * ray.direction;
    float3 color = make_float3(0.f, 0.f, 0.f);
    float3 opacity = p_Ko;
    float3 Kd = p_Kd;

    float3 normal = ::optix::normalize(p_normal);
    const float epsilon = sceneEpsilon * epsilonFactor * optix::length(eye - hit_point);
    if (fmaxf(opacity) > 0.f)
    {
        // User data
        if (simulation_data.size() > 0)
        {
            const float4 userDataColor =
                calcTransferFunctionColor(tfMinValue, tfMinValue + tfRange, simulation_data[simulation_idx], tfColors,
                                          tfOpacities);
            Kd = Kd * (1.f - userDataColor.w) + make_float3(userDataColor) * userDataColor.w;
        }

        const float userParameter = p_user_parameter;

        // Randomness
        optix::size_t2 screen = output_buffer.size();
        unsigned int seed = tea<16>(screen.x * launch_index.y + launch_index.x, frame);

        // Glossiness
        if (p_glossiness < 1.f)
            normal = optix::normalize(normal + (1.f - p_glossiness) *
                                                   make_float3(rnd(seed) - 0.5f, rnd(seed) - 0.5f, rnd(seed) - 0.5f));

        // compute direct lighting
        float3 directLightingColor = make_float3(0.f, 0.f, 0.f);
        unsigned int num_lights = lights.size();
        for (int i = 0; i < num_lights; ++i)
        {
            // Surface
            float light_attenuation = 1.f;

            BasicLight light = lights[i];
            float3 lightDirection;

            if (light.type == BASIC_LIGHT_TYPE_POINT)
            {
                // Point light
                float3 pos = light.pos;
                if (shadows > 0.f && softShadows > 0.f)
                    // Soft shadows
                    pos += softShadows * make_float3(rnd(seed) - 0.5f, rnd(seed) - 0.5f, rnd(seed) - 0.5f);
                lightDirection = optix::normalize(pos - hit_point);
            }
            else
            {
                // Directional light
                lightDirection = -light.pos;
                if (shadows > 0.f && softShadows > 0.f)
                    // Soft shadows
                    lightDirection += softShadows * make_float3(rnd(seed) - 0.5f, rnd(seed) - 0.5f, rnd(seed) - 0.5f);
                lightDirection = optix::normalize(lightDirection);
            }

            float nDl = optix::dot(normal, lightDirection);

            // Shadows
            if (shadows > 0.f)
            {
                if (nDl > 0.f && light.casts_shadow)
                {
                    PerRayData_shadow shadow_prd;
                    shadow_prd.attenuation = make_float3(1.f);
                    optix::Ray shadow_ray(hit_point, lightDirection, shadowRayType, epsilon, giDistance);
                    rtTrace(top_shadower, shadow_ray, shadow_prd);

                    // light_attenuation is zero if completely shadowed
                    light_attenuation -= shadows * (1.f - ::optix::luminance(shadow_prd.attenuation));
                }
            }

            // If not completely shadowed, light the hit point
            if (light_attenuation > 0.f)
            {
                const float3 Lc = light.color * light_attenuation;
                switch (p_shadingMode)
                {
                case MaterialShadingMode::diffuse:
                case MaterialShadingMode::diffuse_transparency:
                case MaterialShadingMode::perlin:
                {
                    float pDl = 1.f;
                    if (p_shadingMode == MaterialShadingMode::perlin)
                    {
                        const float3 point = userParameter * hit_point;
                        const float n1 = 0.25f + 0.75f * optix::clamp(worleyNoise(point, 2.f), 0.f, 1.f);
                        pDl = 1.f - n1;
                        normal.x += 0.5f * n1;
                        normal.y += 0.5f * (0.5f - n1);
                        normal.z += 0.5f * (0.25f - n1);
                        normal = optix::normalize(normal);
                    }

                    // Diffuse
                    directLightingColor += light_attenuation * Kd * nDl * pDl * Lc;
                    const float3 H = optix::normalize(lightDirection - ray.direction);
                    const float nDh = optix::dot(normal, H);
                    if (nDh > 0.f)
                    {
                        // Specular
                        const float power = pow(nDh, p_phong_exp);
                        directLightingColor += p_Ks * power * Lc;
                    }
                    if (p_shadingMode == MaterialShadingMode::diffuse_transparency)
                        opacity *= nDh;
                    break;
                }
                case MaterialShadingMode::cartoon:
                {
                    float cosNL = max(0.f, optix::dot(optix::normalize(eye - hit_point), normal));
                    const uint angleAsInt = cosNL * userParameter;
                    cosNL = (float)angleAsInt / userParameter;
                    directLightingColor += light_attenuation * Kd * cosNL * Lc;
                    break;
                }
                case MaterialShadingMode::basic:
                {
                    const float cosNL = optix::max(0.f, optix::dot(optix::normalize(eye - hit_point), normal));
                    directLightingColor += light_attenuation * Kd * cosNL * Lc;
                    break;
                }
                case MaterialShadingMode::electron:
                case MaterialShadingMode::electron_transparency:
                {
                    float cosNL = max(0.f, optix::dot(optix::normalize(eye - hit_point), normal));
                    cosNL = 1.f - pow(cosNL, userParameter);
                    directLightingColor += light_attenuation * Kd * cosNL * Lc;
                    if (p_shadingMode == MaterialShadingMode::electron_transparency)
                        opacity *= cosNL;
                    break;
                }
                case MaterialShadingMode::checker:
                {
                    const int3 point = make_int3(userParameter * (hit_point + make_float3(1e2f)));
                    const int3 p = make_int3(point.x % 2, point.y % 2, point.z % 2);
                    if ((p.x == p.y && p.z == 1) || (p.x != p.y && p.z == 0))
                        directLightingColor += light_attenuation * Kd;
                    else
                        directLightingColor += light_attenuation * (1.f - Kd);
                    break;
                }
                case MaterialShadingMode::goodsell:
                {
                    const float cosNL = max(0.f, optix::dot(optix::normalize(eye - hit_point), normal));
                    directLightingColor += light_attenuation * Kd * (cosNL > userParameter ? 1.f : 0.5f);
                    break;
                }
                default:
                {
                    directLightingColor += light_attenuation * Kd;
                    break;
                }
                }
            }
        }
        color += directLightingColor;

        // Reflection
        if (fmaxf(p_Kr) > 0.f)
        {
            if (prd_radiance.depth < maxBounces)
            {
                PerRayData_radiance reflected_prd;
                reflected_prd.depth = prd_radiance.depth + 1;

                const float3 R = optix::reflect(ray.direction, normal);
                const optix::Ray reflected_ray(hit_point, R, radianceRayType, epsilon, giDistance);
                rtTrace(top_object, reflected_ray, reflected_prd);
                color = color * (1.f - p_Kr) + p_Kr * reflected_prd.result;
            }
        }

        // Ambient occlusion
        if (giSamples > 0 && giWeight > 0.f)
        {
            float3 aa_color = make_float3(0.f);
            for (int i = 0; i < giSamples; ++i)
            {
                if (prd_radiance.depth >= maxBounces)
                    continue;

                PerRayData_radiance aa_prd;
                aa_prd.depth = prd_radiance.depth + 1;

                float3 aa_normal = optix::normalize(make_float3(rnd(seed) - 0.5f, rnd(seed) - 0.5f, rnd(seed) - 0.5f));
                if (optix::dot(aa_normal, normal) < 0.f)
                    aa_normal = -aa_normal;

                const optix::Ray aa_ray(hit_point, aa_normal, shadowRayType, epsilon, giDistance);
                rtTrace(top_object, aa_ray, aa_prd);
                aa_color = aa_color + giWeight * aa_prd.result;
            }
            color += aa_color / giSamples;
        }

        // Only opaque surfaces are affected by Global Illumination
        if (fmaxf(opacity) == 1.f && prd_radiance.depth < maxBounces)
        {
            // Color bleeding
            if (giWeight > 0.f && prd_radiance.depth == 0)
            {
                PerRayData_radiance new_prd;
                new_prd.depth = prd_radiance.depth + 1;

                float3 ra_normal =
                    ::optix::normalize(make_float3(rnd(seed) - 0.5f, rnd(seed) - 0.5f, rnd(seed) - 0.5f));
                if (optix::dot(ra_normal, normal) < 0.f)
                    ra_normal = -ra_normal;

                const float3 origin = hit_point + epsilonFactor * ra_normal;
                const optix::Ray ra_ray = optix::make_Ray(origin, ra_normal, radianceRayType, epsilon, ray.tmax);
                rtTrace(top_shadower, ra_ray, new_prd);
                color += giWeight * new_prd.result;
            }
        }
    }

    // Refraction
    if (fmaxf(opacity) < 1.f && prd_radiance.depth < maxBounces)
    {
        PerRayData_radiance refracted_prd;
        refracted_prd.depth = prd_radiance.depth + 1;

        const float3 R = refractedVector(ray.direction, normal, p_refractionIndex, 1.f);
        const optix::Ray refracted_ray(hit_point, R, radianceRayType, epsilon, giDistance);
        rtTrace(top_object, refracted_ray, refracted_prd);
        color = color * opacity + (1.f - opacity) * refracted_prd.result;
    }

    float4 finalColor = make_float4(color, fmaxf(opacity));

    // Volume
    if (volumeData.size() > 0)
    {
        const float4 volumeColor = getVolumeContribution(ray);
        compose(volumeColor, finalColor);
    }
    float3 result = make_float3(finalColor);

    // Matrix filter :)
    if (matrixFilter)
        result = make_float3(result.x * 0.666f, result.y * 0.8f, result.z * 0.666f);

    // Exposure and Fog attenuation
    const float z = optix::length(eye - hit_point);
    const float fogAttenuation = z > fogStart ? optix::clamp((z - fogStart) / fogThickness, 0.f, 1.f) : 0.f;
    result = mainExposure * (result * (1.f - fogAttenuation) + fogAttenuation * getEnvironmentColor());

    prd_radiance.result = result;
}

RT_PROGRAM void any_hit_shadow()
{
    phongShadowed(Ko);
}

static __device__ inline void shade(bool textured)
{
    float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));

    float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

    float3 p_Kd = Kd;
    if (textured)
        p_Kd = make_float3(optix::rtTex2D<float4>(albedoMetallic_map, texcoord.x, texcoord.y));

    phongShade(p_Kd, Ka, Ks, Kr, Ko, refraction_index, phong_exp, glossiness, shading_mode, user_parameter, ffnormal);
}

RT_PROGRAM void closest_hit_radiance()
{
    shade(false);
}

RT_PROGRAM void closest_hit_radiance_textured()
{
    shade(true);
}

RT_PROGRAM void exception()
{
    output_buffer[launch_index] = make_color(bad_color);
}
