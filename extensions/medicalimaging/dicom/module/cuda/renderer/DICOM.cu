#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, EPFL/Blue Brain Project
 *
 * The Blue Brain BioExplorer is a tool for scientists to extract and analyse
 * scientific data from visualization
 *
 * This file is part of Blue Brain BioExplorer <https://github.com/BlueBrain/BioExplorer>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <platform/engines/optix6/OptiXCommonStructs.h>
#include <platform/engines/optix6/cuda/Environment.cuh>
#include <platform/engines/optix6/cuda/Helpers.cuh>
#include <platform/engines/optix6/cuda/Random.cuh>

#include <platform/engines/optix6/cuda/renderer/TransferFunction.cuh>

#include <platform/core/common/CommonTypes.h>

using namespace optix;

const float DEFAULT_VOLUME_SHADOW_THRESHOLD = 0.1f;

// System
rtDeclareVariable(float3, bad_color, , );

// Material attributes
rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float3, Kr, , );
rtDeclareVariable(float3, Ko, , );
rtDeclareVariable(float, phong_exp, , );
rtDeclareVariable(uint, shading_mode, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

// Textures
rtDeclareVariable(int, albedoMetallic_map, , );
rtDeclareVariable(float2, texcoord, attribute texcoord, );

// Scene
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );
rtDeclareVariable(unsigned int, frame, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(unsigned int, radianceRayType, , );
rtDeclareVariable(unsigned int, shadowRayType, , );

rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float4, jitter4, , );

// Lights
rtBuffer<BasicLight> lights;
rtDeclareVariable(float3, ambientLightColor, , );

// Volume
rtDeclareVariable(uint3, volumeDimensions, , );
rtDeclareVariable(float3, volumeOffset, , );
rtDeclareVariable(float3, volumeElementSpacing, , );
rtDeclareVariable(uint, volumeSamplesPerRay, , );
rtDeclareVariable(uint, volumeDataTypeSize, , );
rtDeclareVariable(uint, volumeDataType, , );
rtDeclareVariable(int, volumeSampler, , );

// Volume shading
rtDeclareVariable(uint, volumeGradientShadingEnabled, , );
rtDeclareVariable(float, volumeAdaptiveMaxSamplingRate, , );
rtDeclareVariable(uint, volumeSingleShade, , );
rtDeclareVariable(float, volumeSamplingRate, , );
rtDeclareVariable(float3, volumeSpecular, , );
rtDeclareVariable(float, volumeNormalEpsilon, , );

// Transfer function
rtBuffer<float3> tfColors;
rtBuffer<float> tfOpacities;
rtDeclareVariable(float, tfMinValue, , );
rtDeclareVariable(float, tfRange, , );
rtDeclareVariable(uint, tfSize, , );

// Rendering
rtDeclareVariable(float, shadows, , );
rtDeclareVariable(float, softShadows, , );
rtDeclareVariable(float, mainExposure, , );
rtDeclareVariable(float, fogStart, , );
rtDeclareVariable(float, fogThickness, , );
rtDeclareVariable(int, giSamples, , );
rtDeclareVariable(float, giWeight, , );
rtDeclareVariable(float, giDistance, , );

// Clipping planes
rtBuffer<float4, 1> clip_planes;
rtDeclareVariable(unsigned int, nb_clip_planes, , );

rtBuffer<uchar4, 2> output_buffer;

__device__ void applyClippingValues(const optix::Ray& ray, float& near, float& far)
{
    for (int i = 0; i < nb_clip_planes; ++i)
    {
        const float4 clipPlane = clip_planes[i];
        const float3 normal = make_float3(clipPlane.x, clipPlane.y, clipPlane.z);
        float rn = dot(ray.direction, normal);
        if (rn == 0.f)
            rn = 0.01f; // volumeNormalEpsilon;
        const float d = clipPlane.w;
        const float t = -(dot(normal, ray.origin) + d) / rn;
        if (rn > 0.f) // opposite direction plane
            near = max(near, t);
        else
            far = min(far, t);
    }
}

static __device__ inline bool volumeIntersection(const optix::Ray& ray, float& t0, float& t1)
{
    float3 boxmin = volumeOffset + make_float3(0.f);
    float3 boxmax = volumeOffset + make_float3(volumeDimensions) / volumeElementSpacing;

    float3 a = (boxmin - ray.origin) / ray.direction;
    float3 b = (boxmax - ray.origin) / ray.direction;
    float3 near = fminf(a, b);
    float3 far = fmaxf(a, b);
    t0 = fmaxf(near);
    t1 = fminf(far);

    // applyClippingValues(ray, t0, t1);
    return (t0 <= t1);
}

static __device__ void compose(const float4& src, float4& dst, const float alphaCorrection = 1.0)
{
    const float alpha = alphaCorrection * src.w;
    dst =
        make_float4((1.f - dst.w) * alpha * make_float3(src) + dst.w * make_float3(dst), dst.w + alpha * (1.f - dst.w));
}

static __device__ float getVoxelValue(const float3& p)
{
    switch (volumeDataType)
    {
    case RT_FORMAT_BYTE:
    {
        const char voxelValue = optix::rtTex3D<char>(volumeSampler, p.x, p.y, p.z);
        return float(voxelValue) / 256.f;
    }
    case RT_FORMAT_UNSIGNED_BYTE:
    {
        const unsigned char voxelValue = optix::rtTex3D<unsigned char>(volumeSampler, p.x, p.y, p.z);
        return float(voxelValue) / 256.f;
    }
    case RT_FORMAT_INT:
    {
        const int voxelValue = optix::rtTex3D<int>(volumeSampler, p.x, p.y, p.z);
        return float(voxelValue) / 65536.f;
    }
    case RT_FORMAT_UNSIGNED_INT:
    {
        const unsigned int voxelValue = optix::rtTex3D<unsigned int>(volumeSampler, p.x, p.y, p.z);
        return float(voxelValue) / 65536.f;
    }
    default:
    {
        return optix::rtTex3D<float>(volumeSampler, p.x, p.y, p.z);
    }
    }
}

static __device__ float getVolumeShadowContribution(const optix::Ray& volumeRay, const float limit = 1.0)
{
    float shadowIntensity = 0.f;
    float t0, t1;
    if (!volumeIntersection(volumeRay, t0, t1))
        return shadowIntensity;

    applyClippingValues(volumeRay, t0, t1);

    t0 = max(0.f, t0);
    const float tstep = volumeSamplingRate;
    float t = t0 + tstep;
    float distance = 0.f;

    optix::size_t2 screen = output_buffer.size();
    unsigned int seed = tea<16>(screen.x * launch_index.y + launch_index.x, frame);

    while (t < t1 && shadowIntensity < limit && distance < giDistance)
    {
        const float random = rnd(seed) * tstep;
        const float3 point =
            ((volumeRay.origin + volumeSamplingRate + volumeRay.direction * (t + random)) - volumeOffset) /
            volumeElementSpacing;

        // if (!isClipped(point))
        {
            if (point.x > 0.f && point.x < volumeDimensions.x && point.y > 0.f && point.y < volumeDimensions.y &&
                point.z > 0.f && point.z < volumeDimensions.z)
            {
                const float3 p = make_float3(point.x / volumeDimensions.x / 2.f, point.y / volumeDimensions.y / 2.f,
                                             point.z / volumeDimensions.z / 2.f);
                const float4 voxelColor = calcTransferFunctionColor(tfMinValue, tfMinValue + tfRange, getVoxelValue(p),
                                                                    tfColors, tfOpacities);

                shadowIntensity += voxelColor.w;
            }
        }
        t += tstep;
        distance += tstep;
    }
    return shadowIntensity;
}

static __device__ float4 getVolumeContribution(const optix::Ray& volumeRay)
{
    if (tfColors.size() == 0)
        return make_float4(0.f, 1.f, 0.f, 0.f);

    float4 pathColor = make_float4(0.f, 0.f, 0.f, 0.f);

    float t0, t1;
    if (!volumeIntersection(volumeRay, t0, t1))
        return pathColor;

    applyClippingValues(volumeRay, t0, t1);

    optix::size_t2 screen = output_buffer.size();
    unsigned int seed = tea<16>(screen.x * launch_index.y + launch_index.x, frame);

    float t = max(0.f, t0);
    while (t < (t1 - volumeSamplingRate) && pathColor.w < 1.f)
    {
        const float3 absolutePoint = ray.origin + t * ray.direction;
        // if (!isClipped(absolutePoint))
        {
            const float random = rnd(seed) * volumeSamplingRate;
            const float3 point =
                ((volumeRay.origin + volumeSamplingRate + volumeRay.direction * (t + random)) - volumeOffset) /
                volumeElementSpacing;

            float4 voxelColor = make_float4(0.f);
            float shadowIntensity = 0.f;
            float aaIntensity = 0.f;

            if (point.x >= 0.f && point.x < volumeDimensions.x && point.y >= 0.f && point.y < volumeDimensions.y &&
                point.z >= 0.f && point.z < volumeDimensions.z)
            {
                const float3 p = make_float3(point.x / volumeDimensions.x / 2.f, point.y / volumeDimensions.y / 2.f,
                                             point.z / volumeDimensions.z / 2.f);
                voxelColor += calcTransferFunctionColor(tfMinValue, tfMinValue + tfRange, getVoxelValue(p), tfColors,
                                                        tfOpacities);

                // Determine light contribution
                if (shadows > 0.f && voxelColor.w > DEFAULT_VOLUME_SHADOW_THRESHOLD)
                {
                    for (int i = 0; i < lights.size(); ++i)
                    {
                        BasicLight light = lights[i];
                        optix::Ray shadowRay = volumeRay;
                        switch (light.type)
                        {
                        case BASIC_LIGHT_TYPE_POINT:
                        {
                            // Point light
                            float3 lightPosition = light.pos;
                            if (softShadows > 0.f)
                                // Soft shadows
                                lightPosition +=
                                    softShadows * make_float3(rnd(seed) - 0.5f, rnd(seed) - 0.5f, rnd(seed) - 0.5f);
                            shadowRay.origin = lightPosition;
                            shadowRay.direction = optix::normalize(lightPosition - absolutePoint);
                            break;
                        }
                        case BASIC_LIGHT_TYPE_DIRECTIONAL:
                        {
                            // Directional light
                            float3 lightDirection = light.dir;
                            if (softShadows > 0.f)
                                // Soft shadows
                                lightDirection +=
                                    softShadows * make_float3(rnd(seed) - 0.5f, rnd(seed) - 0.5f, rnd(seed) - 0.5f);
                            shadowRay.origin = absolutePoint;
                            shadowRay.direction = -optix::normalize(lightDirection);
                            break;
                        }
                        }

                        shadowIntensity += getVolumeShadowContribution(shadowRay) * shadows;
                    }
                }

                // Ambient occlusion
                for (int i = 0; i < giSamples && voxelColor.w > DEFAULT_VOLUME_SHADOW_THRESHOLD; ++i)
                {
                    const float3 aa_normal =
                        optix::normalize(make_float3(rnd(seed) - 0.5f, rnd(seed) - 0.5f, rnd(seed) - 0.5f));
                    optix::Ray aa_ray = volumeRay;
                    aa_ray.origin = point;
                    aa_ray.direction = aa_normal;
                    aaIntensity += getVolumeShadowContribution(aa_ray) * giWeight;
                }
                if (giSamples > 0)
                    aaIntensity /= float(giSamples);

#if 0
                // Shading
                float3 normal = make_float3(0.f);
                const float3 positions[6] = {
                    {-1, 0, 0}, {1, 0, 0}, {0, -1, 0}, {0, 1, 0}, {0, 0, -1}, {0, 0, 1},
                };
                for (uint i = 0; i < 6; ++i)
                    if (v > getVoxelValue(p + positions[i] * volumeNormalEpsilon))
                        normal = normal + positions[i];
                    else
                        normal = normal - positions[i];

                normal = ::optix::normalize(normal);

                float3 specularColor = make_float3(voxelColor);
                for (int i = 0; i < lights.size(); ++i)
                {
                    BasicLight light = lights[i];
                    const float3 hit_point = ray.origin + t_hit * ray.direction;
                    const float3 L = normalize(light.pos - hit_point);
                    const float d = max(0.f, dot(normal, L));
                    // const float phong_exp = 50.f;
                    // float power = pow(nDl, phong_exp);
                    specularColor = specularColor * d * light.color;
                }
                voxelColor = make_float4(specularColor, voxelColor.w);
#endif
                const float lightAttenuation = 1.f - (shadowIntensity + aaIntensity) * voxelColor.w;
                voxelColor.x *= lightAttenuation;
                voxelColor.y *= lightAttenuation;
                voxelColor.z *= lightAttenuation;
                compose(voxelColor, pathColor);
            }
        }
        t += volumeSamplingRate;
    }

    compose(make_float4(getEnvironmentColor(), 1.f - pathColor.w), pathColor);

    return ::optix::clamp(pathColor, 0.f, 1.f);
}

static __device__ void volumeShadowed(float3 p_Ko)
{
    // this material is opaque, so it fully attenuates all shadow rays
    prd_shadow.attenuation = 1.f - p_Ko;
    rtTerminateRay();
}

static __device__ void volumeShade()
{
    const float4 color = getVolumeContribution(ray);
    float3 result = make_float3(::optix::clamp(color * mainExposure, 0.f, 1.f));

    // Exposure and Fog attenuation
    const float3 hit_point = ray.origin + t_hit * ray.direction;
    const float z = optix::length(eye - hit_point);
    const float fogAttenuation = z > fogStart ? optix::clamp((z - fogStart) / fogThickness, 0.f, 1.f) : 0.f;
    result = (result * (1.f - fogAttenuation) + fogAttenuation * getEnvironmentColor());

    prd_radiance.result = result;
    prd_radiance.importance = 1.f;
    prd_radiance.depth += 1;
}

RT_PROGRAM void any_hit_shadow()
{
    volumeShadowed(Ko);
}

static __device__ inline void shade(bool textured)
{
    volumeShade();
}

RT_PROGRAM void closest_hit_radiance()
{
    shade(false);
}

RT_PROGRAM void closest_hit_radiance_textured()
{
    shade(true);
}

RT_PROGRAM void exception()
{
    output_buffer[launch_index] = make_color(bad_color);
}
