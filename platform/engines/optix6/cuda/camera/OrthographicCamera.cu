#include "hip/hip_runtime.h"
/* Copyright (c) 2019, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 *
 * This file is part of Brayns <https://github.com/BlueBrain/Brayns>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include "../../OptiXCommonStructs.h"

#include "../Helpers.cuh"
#include "../Random.cuh"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, dir, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float3, bad_color, , );
rtDeclareVariable(float, scene_epsilon, , );
rtBuffer<uchar4, 2> output_buffer;
rtBuffer<float4, 2> accum_buffer;
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, frame, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(float, height, , );
rtDeclareVariable(float4, jitter4, , );
rtDeclareVariable(unsigned int, samples_per_pixel, , );

rtBuffer<float4, 1> clip_planes;
rtDeclareVariable(unsigned int, nb_clip_planes, , );

__device__ void getClippingValues(const float3& ray_origin, const float3& ray_direction, float& near, float& far)
{
    for (int i = 0; i < nb_clip_planes; ++i)
    {
        float4 clipPlane = clip_planes[i];
        const float3 planeNormal = {clipPlane.x, clipPlane.y, clipPlane.z};
        float rn = dot(ray_direction, planeNormal);
        if (rn == 0.f)
            rn = scene_epsilon;
        float d = clipPlane.w;
        float t = -(dot(planeNormal, ray_origin) + d) / rn;
        if (rn > 0.f) // opposite direction plane
            near = max(near, t);
        else
            far = min(far, t);
    }
}

// Pass 'seed' by reference to keep randomness state
__device__ float3 launch(unsigned int& seed, const float2 screen, const bool use_randomness)
{
    // Subpixel jitter: send the ray through a different position inside the
    // pixel each time, to provide antialiasing.
    float2 subpixel_jitter = use_randomness ? make_float2(rnd(seed) - 0.5f, rnd(seed) - 0.5f) : make_float2(0.f, 0.f);

    float2 p = (make_float2(launch_index) + subpixel_jitter) / screen * 2.f - 1.f;

    const float3 ray_origin = W + screen.x * U + screen.y * V;
    const float3 ray_direction = optix::normalize(dir);

    PerRayData_radiance prd;
    prd.importance = 1.f;
    prd.depth = 0;
    prd.rayDdx = screen.x * U;
    prd.rayDdy = screen.y * V;

    // lens sampling
    float2 sample = optix::square_to_disk(make_float2(jitter4.z, jitter4.w));

    float near = scene_epsilon;
    float far = INFINITY;

    getClippingValues(ray_origin, ray_direction, near, far);

    optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, near, far);

    rtTrace(top_object, ray, prd);

    return prd.result;
}

RT_PROGRAM void orthographicCamera()
{
    const size_t2 screen = output_buffer.size();
    const float2 screen_f = make_float2(screen);

    unsigned int seed = tea<16>(screen.x * launch_index.y + launch_index.x, frame);

    const int num_samples = max(1, samples_per_pixel);
    // We enable randomness if we are using subpixel sampling or accumulation
    const bool use_randomness = frame > 0 || num_samples > 1;

    float3 result = make_float3(0, 0, 0);
    for (int i = 0; i < num_samples; i++)
        result += launch(seed, screen_f, use_randomness);
    result /= num_samples;

    float4 acc_val;
    if (frame > 0)
    {
        acc_val = accum_buffer[launch_index];
        acc_val = lerp(acc_val, make_float4(result, 0.f), 1.0f / static_cast<float>(frame + 1));
    }
    else
        acc_val = make_float4(result, 1.f);

    output_buffer[launch_index] = make_color(make_float3(acc_val));

    if (accum_buffer.size().x > 1 && accum_buffer.size().y > 1)
        accum_buffer[launch_index] = acc_val;
}

RT_PROGRAM void exception()
{
    output_buffer[launch_index] = make_color(bad_color);
}
