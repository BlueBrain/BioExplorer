#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 *
 * This file is part of Blue Brain BioExplorer <https://github.com/BlueBrain/BioExplorer>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <platform/engines/optix6/cuda/Context.cuh>
#include <platform/engines/optix6/cuda/Helpers.cuh>
#include <platform/engines/optix6/cuda/Random.cuh>

using namespace optix;

// Pass 'seed' by reference to keep randomness state
__device__ float4 launch(uint& seed, const float2 screen, const bool use_randomness)
{
    // Subpixel jitter: send the ray through a different position inside the
    // pixel each time, to provide anti-aliasing.
    const float2 subpixel_jitter =
        use_randomness ? make_float2(rnd(seed) - 0.5f, rnd(seed) - 0.5f) : make_float2(0.f, 0.f);

    // Normalized pixel position (from -0.5 to 0.5)
    const float2 p = (make_float2(launch_index) + subpixel_jitter) / screen * 2.f - 1.f;

    const float3 ray_origin = W + p.x * U + p.y * V;
    const float3 ray_direction = dir;

    PerRayData_radiance prd;
    prd.importance = 1.f;
    prd.depth = 0;
    prd.zDepth = INFINITY;

    float near = sceneEpsilon;
    float far = INFINITY;

    // Clipping planes
    if (enableClippingPlanes)
        applyClippingPlanes(ray_origin, ray_direction, near, far);

    const optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, near, far);
    rtTrace(top_object, ray, prd);

    return make_float4(make_float3(prd.result) * mainExposure, prd.result.w);
}

RT_PROGRAM void orthographicCamera()
{
    const size_t2 screen = output_buffer.size();
    const float2 screen_f = make_float2(screen);

    uint seed = tea<16>(screen.x * launch_index.y + launch_index.x, frame);

    const int num_samples = max(1, samples_per_pixel);
    // We enable randomness if we are using subpixel sampling or accumulation
    const bool use_randomness = frame > 0 || num_samples > 1;
    float4 result = make_float4(0.f);
    for (int i = 0; i < num_samples; i++)
        result += launch(seed, screen_f, use_randomness);
    result /= num_samples;

    float4 acc_val;
    if (frame > 0)
    {
        acc_val = accum_buffer[launch_index];
        acc_val = lerp(acc_val, result, 1.0f / static_cast<float>(frame + 1));
    }
    else
        acc_val = result;

    output_buffer[launch_index] = make_color(acc_val);

    if (accum_buffer.size().x > 1 && accum_buffer.size().y > 1)
        accum_buffer[launch_index] = acc_val;
}

RT_PROGRAM void exception()
{
    output_buffer[launch_index] = make_color(bad_color);
    depth_buffer[launch_index] = INFINITY;
}
