#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 *
 * This file is part of Blue Brain BioExplorer <https://github.com/BlueBrain/BioExplorer>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <platform/engines/optix6/cuda/Environment.cuh>
#include <platform/engines/optix6/cuda/Helpers.cuh>
#include <platform/engines/optix6/cuda/Random.cuh>
#include <platform/engines/optix6/cuda/renderer/Volume.cuh>

#include <platform/core/common/CommonTypes.h>

static __device__ inline void shade(bool textured)
{
    const float3 world_shading_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    const float3 world_geometric_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
    const float3 normal = optix::faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

    float3 color;
    if (textured && albedoMetallic_map)
        color = make_float3(optix::rtTex2D<float4>(albedoMetallic_map, texcoord.x, texcoord.y));
    else
        color = Kd;

    prd.result = make_float4(color * max(0.f, optix::dot(-ray.direction, normal)), 1.f);
}

RT_PROGRAM void any_hit_shadow()
{
    rtTerminateRay();
}

RT_PROGRAM void closest_hit_radiance()
{
    shade(false);
}

RT_PROGRAM void closest_hit_radiance_textured()
{
    shade(true);
}

RT_PROGRAM void exception()
{
    output_buffer[launch_index] = make_color(bad_color);
}
