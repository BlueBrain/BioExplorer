#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 *
 * This file is part of Blue Brain BioExplorer <https://github.com/BlueBrain/BioExplorer>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <platform/engines/optix6/cuda/Environment.cuh>
#include <platform/engines/optix6/cuda/Helpers.cuh>
#include <platform/engines/optix6/cuda/Random.cuh>
#include <platform/engines/optix6/cuda/renderer/Volume.cuh>

#include <platform/core/common/CommonTypes.h>

static __device__ inline void shade(bool textured)
{
    float3 world_shading_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 world_geometric_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));

    float3 p_normal = optix::faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

    float3 p_Kd;
    if (textured && albedoMetallic_map)
        p_Kd = make_float3(optix::rtTex2D<float4>(albedoMetallic_map, texcoord.x, texcoord.y));
    else
        p_Kd = Kd;

    if (simulation_data.size() > 0)
    {
        const float4 userDataColor =
            calcTransferFunctionColor(transfer_function_map, value_range, simulation_data[simulation_idx]);
        p_Kd = p_Kd * (1.f - userDataColor.w) + make_float3(userDataColor) * userDataColor.w;
    }

    prd.result = make_float4(p_Kd * max(0.f, optix::dot(-ray.direction, p_normal)), 1.f);
}

RT_PROGRAM void any_hit_shadow()
{
    rtTerminateRay();
}

RT_PROGRAM void closest_hit_radiance()
{
    shade(false);
}

RT_PROGRAM void closest_hit_radiance_textured()
{
    shade(true);
}

RT_PROGRAM void exception()
{
    output_buffer[launch_index] = make_color(bad_color);
}
