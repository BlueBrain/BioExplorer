#include "hip/hip_runtime.h"
/*
 *
 * The Blue Brain BioExplorer is a tool for scientists to extract and analyse
 * scientific data from visualization
 *
 * This file is part of Blue Brain BioExplorer <https://github.com/BlueBrain/BioExplorer>
 *
 * Copyright 2020-2023 Blue BrainProject / EPFL
 *
 * This program is free software: you can redistribute it and/or modify it under
 * the terms of the GNU General Public License as published by the Free Software
 * Foundation, either version 3 of the License, or (at your option) any later
 * version.
 *
 * This program is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more
 * details.
 *
 * You should have received a copy of the GNU General Public License along with
 * this program.  If not, see <https://www.gnu.org/licenses/>.
 */

// Ray-cone intersection: based on Ching-Kuang Shene (Graphics Gems 5, p. 227-230)

#include <platform/engines/optix6/cuda/Context.cuh>

// Global variables
rtDeclareVariable(uint, sdf_geometry_size, , );
rtBuffer<uint8_t> sdf_geometries_buffer;
rtBuffer<uint64_t> sdf_geometries_neighbours_buffer;

#define SDF_NO_INTERSECTION -1.f

enum SDFType : uint8_t
{
    Sphere = 0,
    Pill = 1,
    ConePill = 2,
    ConePillSigmoid = 3,
    Cone = 4
};

#define OFFSET_USER_DATA 0
#define OFFSET_USER_PARAMS (OFFSET_USER_DATA + sizeof(uint64_t))
#define OFFSET_P0 (OFFSET_USER_PARAMS + sizeof(float3))
#define OFFSET_P1 (OFFSET_P0 + sizeof(float3))
#define OFFSET_R0 (OFFSET_P1 + sizeof(float3))
#define OFFSET_R1 (OFFSET_R0 + sizeof(float))
#define OFFSET_NEIGHBOUR_INDEX (OFFSET_R1 + sizeof(float))
#define OFFSET_NB_NEIGHBOURS (OFFSET_NEIGHBOUR_INDEX + sizeof(uint64_t))
#define OFFSET_TYPE (OFFSET_NB_NEIGHBOURS + sizeof(uint8_t))

//////////////////////////////////////////////////////////////////////

// NOTE: This layout must match exactly the 'SDFGeometry' struct in
// 'SDFGeometry.h'
struct SDFGeometry
{
    uint64_t userData;
    float3 userParams;
    float3 p0;
    float3 p1;
    float r0;
    float r1;
    uint64_t neighboursIndex;
    uint8_t numNeighbours;
    uint8_t type;
};

#if 0
rtBuffer<uint64_t> sdf_geometries_indices_buffer;

static __device__ inline float mix(const float x, const float y, const float a)
{
    return x * (1.f - a) + y * a;
}

// https://en.wikipedia.org/wiki/Smoothstep
static __device__ inline float smootherStep(const float x)
{
    return x * x * x * (x * (x * 6.f - 15.f) + 10.f);
}
#endif

static __device__ inline float sign(const float x)
{
    return (x >= 0.f ? 1.f : -1.f);
}

static __device__ inline float lerp(const float factor, const float a, const float b)
{
    return (1.f - factor) * a + factor * b;
}

// polynomial smooth min (k = 0.1);
static __device__ inline float sminPoly(const float a, const float b, const float k)
{
    const float h = ::optix::clamp(0.5f + 0.5f * (b - a) / k, 0.f, 1.f);
    return lerp(h, b, a) - k * h * (1.f - h);
}

static __device__ inline float opDisplacement(const float3& p, const float a, const float b)
{
    return a * sin(b * p.x) * sin(b * p.y * 0.6f) * sin(b * p.z * 0.3f);
}

static __device__ inline float sdSphere(const float3& p, const float3& c, float r)
{
    return ::optix::length(p - c) - r;
}

static __device__ inline float sdCapsule(const float3& p, const float3& a, const float3& b, const float r)
{
    const float3 pa = p - a;
    const float3 ba = b - a;
    const float h = ::optix::clamp(::optix::dot(pa, ba) / ::optix::dot(ba, ba), 0.f, 1.f);
    return ::optix::length(pa - ba * h) - r;
}

static __device__ inline float sdConePill(const float3& p, const float3& a, const float3& b, const float r1,
                                          const float r2, const bool /*useSigmoid*/)
{
    // sampling independent computations (only depend on shape)
    const float3 ba = b - a;
    float l2 = ::optix::dot(ba, ba);
    float rr = r1 - r2;
    float a2 = l2 - rr * rr;
    float il2 = 1.0 / l2;

    // sampling dependant computations
    const float3 pa = p - a;
    const float y = ::optix::dot(pa, ba);
    const float z = y - l2;
    const float3 d = pa * l2 - ba * y;
    const float x2 = ::optix::dot(d, d);
    const float y2 = y * y * l2;
    const float z2 = z * z * l2;

    // single square root!
    const float k = sign(rr) * rr * rr * x2;
    if (sign(z) * a2 * z2 > k)
        return sqrt(x2 + z2) * il2 - r2;
    if (sign(y) * a2 * y2 < k)
        return sqrt(x2 + y2) * il2 - r1;
    return (sqrt(x2 * a2 * il2) + y * rr) * il2 - r1;
}

static __device__ inline float sdCone(const float3& p, const float3 a, const float3 b, float ra, float rb)
{
    float rba = rb - ra;
    float baba = ::optix::dot(b - a, b - a);
    float papa = ::optix::dot(p - a, p - a);
    float paba = ::optix::dot(p - a, b - a) / baba;

    float x = sqrt(papa - paba * paba * baba);

    float cax = max(0.0, x - ((paba < 0.5) ? ra : rb));
    float cay = abs(paba - 0.5) - 0.5;

    float k = rba * rba + baba;
    float f = ::optix::clamp((rba * (x - ra) + paba * baba) / k, 0.0, 1.0);

    float cbx = x - ra - f * rba;
    float cby = paba - f;

    float s = (cbx < 0.0 && cay < 0.0) ? -1.0 : 1.0;

    return s * sqrt(min(cax * cax + cay * cay * baba, cbx * cbx + cby * cby * baba));
}

static __device__ inline bool intersectBox(const ::optix::Aabb& box, float& t0, float& t1)
{
    const float3 a = (box.m_min - ray.origin) / ray.direction;
    const float3 b = (box.m_max - ray.origin) / ray.direction;
    const float3 near = fminf(a, b);
    const float3 far = fmaxf(a, b);
    t0 = fmaxf(near);
    t1 = fminf(far);

    return (t0 <= t1);
}

static __device__ inline SDFGeometry getPrimitive(const int primIdx)
{
    SDFGeometry primitive;
    const uint64_t idx = primIdx * sdf_geometry_size;
    memcpy(&primitive, &sdf_geometries_buffer[idx], sdf_geometry_size);
    return primitive;
}

static __device__ inline uint64_t getNeighbourIdx(const uint64_t startIdx, const uint8_t neighIdx)
{
    return sdf_geometries_neighbours_buffer[startIdx + neighIdx];
}

static __device__ inline ::optix::Aabb getBounds(const SDFGeometry& primitive)
{
    const float radius = max(primitive.r0, primitive.r1) + primitive.userParams.x;
    ::optix::Aabb aabb;
    if (primitive.type == SDFType::Sphere)
    {
        aabb.m_min = primitive.p0 - radius;
        aabb.m_max = primitive.p0 + radius;
        return aabb;
    }
    aabb.m_min = make_float3(min(primitive.p0.x, primitive.p1.x), min(primitive.p0.y, primitive.p1.y),
                             min(primitive.p0.z, primitive.p1.z)) -
                 radius;
    aabb.m_max = make_float3(max(primitive.p0.x, primitive.p1.x), max(primitive.p0.y, primitive.p1.y),
                             max(primitive.p0.z, primitive.p1.z)) +
                 radius;
    return aabb;
}

//////////////////////////////////////////////////////////////////////

static __device__ inline float calcDistance(const SDFGeometry& primitive, const float3& position,
                                            const bool processDisplacement)
{
    const float displacement = (processDisplacement && primitive.userParams.x > 0.f)
                                   ? opDisplacement(position, primitive.userParams.x, primitive.userParams.y)
                                   : 0.f;
    if (primitive.type == SDFType::Sphere)
        return displacement + sdSphere(position, primitive.p0, primitive.r0);
    if (primitive.type == SDFType::Pill)
        return displacement + sdCapsule(position, primitive.p0, primitive.p1, primitive.r0);
    if (primitive.type == SDFType::ConePill || primitive.type == SDFType::ConePillSigmoid)
        return displacement + sdConePill(position, primitive.p0, primitive.p1, primitive.r0, primitive.r1,
                                         primitive.type == SDFType::ConePillSigmoid);
    if (primitive.type == SDFType::Cone)
        return displacement + sdCone(position, primitive.p0, primitive.p1, primitive.r0, primitive.r1);
    return SDF_NO_INTERSECTION; // TODO: Weird return value...
}

//////////////////////////////////////////////////////////////////////

static __device__ inline float sdfDistance(const float3& position, const SDFGeometry& primitive,
                                           const bool processDisplacement)
{
    float d = calcDistance(primitive, position, processDisplacement);

    if (processDisplacement && primitive.numNeighbours > 0)
    {
        const float l1 = ::optix::length(primitive.p0 - position);
        const float l2 = ::optix::length(primitive.p1 - position);
        const float r0 = max(primitive.r0, primitive.r1);

        for (uint8_t i = 0; i < primitive.numNeighbours; ++i)
        {
            const uint64_t neighbourIndex = getNeighbourIdx(primitive.neighboursIndex, i);

            const SDFGeometry neighbourGeometry = getPrimitive(neighbourIndex);

            const float dOther = calcDistance(neighbourGeometry, position, processDisplacement);
            if (dOther < 0.f)
                continue;
            const float l1 = ::optix::length(neighbourGeometry.p0 - position);
            const float l2 = ::optix::length(neighbourGeometry.p1 - position);
            const float r1 = max(neighbourGeometry.r0, neighbourGeometry.r1);
            const float blendFactor = lerp(geometrySdfBlendLerpFactor, min(r0, r1), max(r0, r1));

            d = sminPoly(dOther, d, blendFactor * geometrySdfBlendFactor);
        }
    }
    return d;
}

//////////////////////////////////////////////////////////////////////

static __device__ inline float3 computeNormal(const float3& position, const SDFGeometry& primitive,
                                              const bool processDisplacement)
{
    // tetrahedron technique (4 evaluations)
    const float t = 0.1f;
    const float3 k0 = make_float3(t, -t, -t);
    const float3 k1 = make_float3(-t, -t, t);
    const float3 k2 = make_float3(-t, t, -t);
    const float3 k3 = make_float3(t, t, t);
    return ::optix::normalize(k0 * sdfDistance(position + geometrySdfEpsilon * k0, primitive, processDisplacement) +
                              k1 * sdfDistance(position + geometrySdfEpsilon * k1, primitive, processDisplacement) +
                              k2 * sdfDistance(position + geometrySdfEpsilon * k2, primitive, processDisplacement) +
                              k3 * sdfDistance(position + geometrySdfEpsilon * k3, primitive, processDisplacement));
}

/////////////////////////////////////////////////////////////////////////////

static __device__ inline float rayMarching(const SDFGeometry& primitive, bool& processDisplacement)
{
    const ::optix::Aabb box = getBounds(primitive);

    float t0, t1;
    if (!intersectBox(box, t0, t1))
        return SDF_NO_INTERSECTION;

    // TODO compute pixel radius
    const float pixel_radius = geometrySdfEpsilon;

    float omega = geometrySdfOmega;
    float t = t0;
    float candidateError = 1e6f;
    float tCandidate = t0;
    float previousRadius = 0.f;
    float stepLength = 0.f;
    uint64_t stepCount = 0;
    const bool forceHit = true;

    // check if we start inside or outside of the shape
    const float sdfSign = (sdfDistance(ray.origin, primitive, true) < 0.f ? -1 : 1);

    for (uint64_t i = 0; i < geometrySdfNbMarchIterations; i++)
    {
        const float3 p = ray.origin + ray.direction * t;
        processDisplacement = (/*ray.flags == RAY_FLAG_PRIMARY && */ ::optix::length(p - eye) < geometrySdfDistance);

        float signed_radius = sdfSign * sdfDistance(p, primitive, processDisplacement);
        float radius = abs(signed_radius);
        bool sorFail = (omega > 1.f && (radius + previousRadius) < stepLength);

        if (sorFail)
        {
            stepLength -= omega * stepLength;
            omega = 1.f;
        }
        else
            stepLength = signed_radius * omega;

        previousRadius = radius;
        float error = radius / t;
        if (!sorFail && error < candidateError)
        {
            tCandidate = t;
            candidateError = error;
        }

        if (!sorFail && (error < pixel_radius || t > t1))
            break;

        t += stepLength;
        ++stepCount;
    }

    if (t > t1 || (candidateError > pixel_radius && !forceHit))
        return SDF_NO_INTERSECTION;

    return tCandidate;
}

template <bool use_robust_method>
static __device__ void intersect_sdf_geometry(int primIdx)
{
    const SDFGeometry primitive = getPrimitive(primIdx);
    bool processDisplacement = true;
    const float t_in = rayMarching(primitive, processDisplacement);

    if (t_in > 0.f)
    {
        rtPotentialIntersection(t_in);
        if (t_in > ray.tmin && t_in < ray.tmax)
        {
            const float3 position = ray.origin + t_in * ray.direction;
            shading_normal = geometric_normal = computeNormal(position, primitive, processDisplacement);
            userDataIndex = primitive.userData;
            texcoord = make_float2(0.f);
            texcoord3d = make_float3(0.f);
            rtReportIntersection(0);
        }
    }
}

RT_PROGRAM void intersect(int primIdx)
{
    intersect_sdf_geometry<false>(primIdx);
}

RT_PROGRAM void robust_intersect(int primIdx)
{
    intersect_sdf_geometry<true>(primIdx);
}

RT_PROGRAM void bounds(int primIdx, float result[6])
{
    const SDFGeometry primitive = getPrimitive(primIdx);
    const ::optix::Aabb bounds = getBounds(primitive);
    memcpy(&result[0], &bounds[0], sizeof(optix::Aabb));
}
