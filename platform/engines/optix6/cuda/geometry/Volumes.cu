#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2015-2023, EPFL/Blue Brain Project
 * All rights reserved. Do not distribute without permission.
 * Responsible Author: Cyrille Favreau <cyrille.favreau@epfl.ch>
 * Author: Jafet Villafranca Diaz <jafet.villafrancadiaz@epfl.ch>
 *
 * Ray-cone intersection:
 * based on Ching-Kuang Shene (Graphics Gems 5, p. 227-230)
 *
 * This file is part of Blue Brain BioExplorer <https://github.com/BlueBrain/BioExplorer>
 *
 * This library is free software; you can redistribute it and/or modify it under
 * the terms of the GNU Lesser General Public License version 3.0 as published
 * by the Free Software Foundation.
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 * details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with this library; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <platform/engines/optix6/cuda/Random.cuh>
#include <platform/engines/optix6/cuda/renderer/Volume.cuh>

using namespace optix;

const uint OFFSET_DIMENSIONS = 0;
const uint OFFSET_POSITION = OFFSET_DIMENSIONS + 3;
const uint OFFSET_SPACING = OFFSET_POSITION + 3;
const uint OFFSET_TEXTURE_SAMPLER_ID = OFFSET_SPACING + 3;

rtDeclareVariable(unsigned int, volume_size, , );

rtBuffer<float> volumes;

template <bool use_robust_method>
static __device__ void intersect_volume(int primIdx)
{
    const int idx = primIdx * volume_size;
    const float3 dimensions = {volumes[idx + OFFSET_DIMENSIONS], volumes[idx + OFFSET_DIMENSIONS + 1],
                               volumes[idx + OFFSET_DIMENSIONS + 2]};
    const float3 position = {volumes[idx + OFFSET_POSITION], volumes[idx + OFFSET_POSITION + 1],
                             volumes[idx + OFFSET_POSITION + 2]};
    const float3 spacing = {volumes[idx + OFFSET_SPACING], volumes[idx + OFFSET_SPACING + 1],
                            volumes[idx + OFFSET_SPACING + 2]};

    const int textureSamplerId = static_cast<int>(volumes[idx + OFFSET_TEXTURE_SAMPLER_ID]);

    const float3 boxMin = position;
    const float3 boxMax = position + dimensions * spacing;

    const float3 a = (boxMin - ray.origin) / ray.direction;
    const float3 b = (boxMax - ray.origin) / ray.direction;
    const float3 near = fminf(a, b);
    const float3 far = fmaxf(a, b);
    float t0 = fmaxf(near);
    float t1 = fminf(far);

    const ::optix::size_t2 screen = output_buffer.size();
    unsigned int seed = tea<16>(screen.x * launch_index.y + launch_index.x, frame);

    const float diag = min(spacing.x, min(spacing.y, spacing.z));
    const float step = diag / volumeSamplingRate;
    const float random = rnd(seed) * step;

    // Apply ray clipping
    t0 = max(t0, ray.tmin);
    t1 = min(t1, ray.tmax);

    if (t0 > 0.f && t0 <= t1)
    {
        float t = t0 + random;
        while (t < t1)
        {
            const float3 p0 = ((ray.origin + t * ray.direction) - position) / (spacing * dimensions);
            const float voxelValue = optix::rtTex3D<float>(textureSamplerId, p0.x, p0.y, p0.z);
            const float4 voxelColor =
                calcTransferFunctionColor(tfMinValue, tfMinValue + tfRange, voxelValue, tfColors, tfOpacities);
            if (voxelColor.w > 0.f)
                if (rtPotentialIntersection(t + step))
                {
                    float3 normal = make_float3(0.f);
                    if (volumeGradientShadingEnabled)
                    {
                        const float3 positions[6] = {{-1, 0, 0}, {1, 0, 0},  {0, -1, 0},
                                                     {0, 1, 0},  {0, 0, -1}, {0, 0, 1}};
                        for (const auto& position : positions)
                        {
                            const float3 p1 = p0 + (position * DEFAULT_GRADIENT_OFFSET);
                            const float v = optix::rtTex3D<float>(textureSamplerId, p1.x, p1.y, p1.z);
                            // if (v > DEFAULT_VOLUME_SHADING_THRESHOLD)
                            normal += v * position;
                        }
                        normal = ::optix::normalize(-1.f * normal);
                    }
                    else
                        normal = make_float3(0, 1, 0);
                    geometric_normal = shading_normal = normal;
                    simulation_idx = 0;
                    texcoord = make_float2(0, 0);
                    texcoord3d = p0;
                    rtReportIntersection(0);
                    break;
                }
            t += step;
        }
    }
}

RT_PROGRAM void intersect(int primIdx)
{
    intersect_volume<false>(primIdx);
}

RT_PROGRAM void robust_intersect(int primIdx)
{
    intersect_volume<true>(primIdx);
}

RT_PROGRAM void bounds(int primIdx, float result[6])
{
    const int idx = primIdx * volume_size;
    const float3 dimensions = {volumes[idx + OFFSET_DIMENSIONS], volumes[idx + OFFSET_DIMENSIONS + 1],
                               volumes[idx + OFFSET_DIMENSIONS + 2]};
    const float3 position = {volumes[idx + OFFSET_POSITION], volumes[idx + OFFSET_POSITION + 1],
                             volumes[idx + OFFSET_POSITION + 2]};
    const float3 spacing = {volumes[idx + OFFSET_SPACING], volumes[idx + OFFSET_SPACING + 1],
                            volumes[idx + OFFSET_SPACING + 2]};

    optix::Aabb* aabb = (optix::Aabb*)result;
    aabb->m_min = position;
    aabb->m_max = position + dimensions * spacing;
}
