#include "hip/hip_runtime.h"
/*
 * The Blue Brain BioExplorer is a tool for scientists to extract and analyse
 * scientific data from visualization
 *
 * Copyright 2020-2023 Blue BrainProject / EPFL
 *
 * This program is free software: you can redistribute it and/or modify it under
 * the terms of the GNU General Public License as published by the Free Software
 * Foundation, either version 3 of the License, or (at your option) any later
 * version.
 *
 * This program is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more
 * details.
 *
 * You should have received a copy of the GNU General Public License along with
 * this program.  If not, see <https://www.gnu.org/licenses/>.
 */

#include <optix_world.h>

#include <platform/engines/optix6/OptiXCommonStructs.h>

// Scene
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );

rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

static __device__ inline void shade()
{
    prd.result = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
}

RT_PROGRAM void any_hit_shadow()
{
    rtTerminateRay();
}

RT_PROGRAM void closest_hit_radiance()
{
    shade();
}
